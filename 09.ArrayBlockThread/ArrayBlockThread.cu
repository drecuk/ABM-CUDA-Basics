#include "hip/hip_runtime.h"
//	##########################################################
//	By Eugene Ch'ng | www.complexity.io
//	Email: genechng@gmail.com
//	----------------------------------------------------------
//  The ERC 'Lost Frontiers' Project
//  Development for the Parallelisation of ABM Simulation
//	----------------------------------------------------------
//	A Basic CUDA Application for ABM Development
//
//  Filling arrays with block generated IDs
//  This application uses both blocks and threads
//  to generate arbitrarily unsigned long numbers
//
//  Using a combination of blocks and threads for this GPU (GTX 750M)
//  we are able to generate a number with 12 zeros - about 4.3 trillion
//  This should be sufficient for a large population of agents!
//  Even larger numbers can be generated using dim3 types for blocks and threads
//  or by incrementing thread IDs after a kernel operation (discussed later)
//
//  The GPU (GTX 750M) used for this development has a limit below:
//  blockIdx = {0 ... 65535}
//  blockDim = 65535
//  threadIdx = {0 ... 1024}
//  65535 * 65535 * 1024 = 4,397,912,294,400
//
//  4 trillion thread IDs should be sufficient for ABM simulation
//  But there are also alternate ways to increase IDs for arbitrarily long arrays
//  Besides limits with blocks*threads, memory is also an issue (see below)
//
//  ----------------------------------------------------------
//  How to compile:
//  nvcc <filename>.cu -o <outputfile>
//	##########################################################
#include <stdio.h>
#include <iostream>
#include "book.h"
using namespace std;

// N = 0.22 billion (1 billion = 9 zeros)
// here we are able to reach 220,000,000+ unsigned integers with the memory
// available (GTX 750M device global memory = 2,097,086,464 = 2GB)
// unsigned long int is 8 byte, 8 x 220,000,000 = 1,760,000,000 bytes = 1.76GB
// this leaves some memory for processing the kernel code.
// putting anything larger will yield invalid argument error using hipMemcpy
// #define N 500000000
const unsigned long int N = 220000000; //200000000

// THREADMAX is measured from using 01.DeviceInfo for GTX 750M
#define THREADMAX 1024

// --------------------- CUDA KERNELS
// Fill arrays with device thread IDs
__global__ void fillArray(unsigned long int *dev_arr)
{
  // we allow blocks and threads to cooperate in generating unsigned long numbers
  // the code below linearise the block and threads into tid used for unsigned long arrays
  // we were previously limited by the thread (1024) and blocks (65535) available
  // to the current GPU (NVIDIA GTX 750M) used for preparing this code
  // using the code below, we can generate
  // threadIdx.x and blockIdx.x is incremental
  // blockDim.x is constant calculated with (N + (THREADMAX-1)/THREADMAX) = 488282.25
  unsigned long int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// assign the dev_array element with tid
  // until it reaches N
  if(tid < N)
  {
      dev_arr[tid] = tid;
  }

}

int main(int argc, const char * argv[])
{
	cout << "------------ initialising device and host arrays" << endl;
  // declaring the array on the stack would cause a segmentation fault
  // as program stack has a limit
  // otherwise, declare the array "unsigned long int arr[N]" outside main
  // to assign it as a global variable - globals are in the heap
  // unsigned long int arr[N];				// host variable

  // here we are instantiating the array on the heap for long int within main
  unsigned long int *arr;
  arr = (unsigned long int*)malloc(N*sizeof(unsigned long int));

  // cout << "-- passed malloc" << endl;

	unsigned long int *dev_arr;  		// device variable
  for(int i=0; i<N; i++)
	{
    // cout << "-- start: " << i << endl;
		arr[i] = 0;

    // commented so we don't need to print all the way up to large values
		// printf("host arr[%d] = %d\n", i, arr[i]);
	}
  size_t s = sizeof(unsigned long int);
  cout << "size of arr: " << s*N << " bytes" << endl;


  cout << "** the last second item of arr[N-1] is:" << "" << arr[N-1] << endl;

	cout << "------------ allocate device memory dev_arr" << endl;
  // allocating a device array to copy to
	// note the N * sizeof(int)
	hipMalloc( (void**)&dev_arr, N * sizeof(unsigned long int) );

	cout << "------------ copy arr to dev_arr" << endl;
  // copying host array to device
  // note the N * sizeof(int)
  size_t size = N * sizeof(unsigned long int);
  cout << size << endl;
	HANDLE_ERROR( hipMemcpy(dev_arr, arr, size, hipMemcpyHostToDevice));

  cout << "------------ calling kernel fillArray" << endl;
  // What's happening here?
  // what we are doing here is to determine the number of blocks needed, in
  // combination with the thread, to generate thread IDs larger than N
  // Let's say we are using threads = 128, with N = 1000 elements
  // (N + (threads-1)/threads = 8.8
  // we will need 8.8 blocks to generate a number > N, adequate for the num ber of
  // thread IDs needed for an array of N size (8.8 * 128 = 1126.4)
  fillArray<<<(unsigned long int)(N + (THREADMAX-1))/THREADMAX,THREADMAX>>>(dev_arr);

  cout << "------------ copy dev_arr to arr" << endl;
	// note the N * sizeof(int)

	HANDLE_ERROR( hipMemcpy(arr, dev_arr, s, hipMemcpyDeviceToHost));

  cout << "------------ printing changed host array" << endl;
	for(unsigned long int i=0; i<N; i++)
	{
    // we want to print only 0-9 and the last 10 values of N
    if(i < 10 || i > N-10)
		  printf("** changed host arr[%ld] = %ld\n", i, arr[i]);
	}

  // ---- FREE ALLOCATED KERNEL MEMORY
	hipFree( dev_arr );
  free(arr);

  return 0;
}
