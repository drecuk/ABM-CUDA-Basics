//	##########################################################
//	By Eugene Ch'ng | www.complexity.io
//	Email: genechng@gmail.com
//	----------------------------------------------------------
//  The ERC 'Lost Frontiers' Project
//  Development for the Parallelisation of ABM Simulation
//	----------------------------------------------------------
//	A Basic CUDA Application for ABM Development
//
//  Filling arrays with thread generated IDs
//  identify a specific thread ID and make changes for that kernel
//
//  LIMITS OF THREADS AND BLOCKS (use 01.DeviceInfo to check your GPU)
//  The particular GPU used here has 1024 threads per block
//  This presents a limit, but we can also use blocks per grid
//  Each block (for this old AlienWare GPU) has 65535 blocks per grid
//  Blocks and Threads have 3 dimensions (type dim3)
//  We will explore how to combine both blocks and threads to create
//  arbitrarily long numbers
//  ----------------------------------------------------------
//  How to compile:
//  nvcc <filename>.cu -o <outputfile>
//	##########################################################

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

// as threads have a size limit of 1024, setting N > 1024 will cause an error
#define N 10

// --------------------- CUDA KERNELS
// Fill arrays with device thread IDs
__global__ void fillArray(int *dev_arr)
{
  // note that we no longer use the for loop here
  // threadIdx.x is a device variable
  // fillArray kernel is called for each thread and has its own ID
  // you can also use threadIdx.y and threadIdx.z if the kernel Chevron Syntax
  // uses dim3 as input
  int tid = threadIdx.x;

	// assign the dev_array element with threadIDx.x
	dev_arr[tid] = tid;

  // identifying a threads
  if(tid == 5)
  {
    printf("**threadIdx.x 5 is called!!\n");
    dev_arr[tid] = tid + 100;
  }
}

// the main is a host code
int main(int argc, const char * argv[])
{
	cout << "------------ initialising device and host arrays" << endl;
  int arr[N];				// host variable
	// int *arr;
  // arr = (int*)malloc(N*sizeof(int));

	int *dev_arr;  		// device variable
	for(int i=0; i<N; i++)
	{
		arr[i] = 0;
		printf("host arr[%d] = %d\n", i, arr[i]);
	}

	cout << "------------ allocate device memory dev_arr" << endl;
  // allocating a device array to copy to
	// note the N * sizeof(int)
	hipMalloc( (void**)&dev_arr, N * sizeof(int) );

	cout << "------------ copy arr to dev_arr" << endl;
  // copying host array to device
  // note the N * sizeof(int)
	hipMemcpy(dev_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);

  cout << "------------ calling kernel fillArray" << endl;
  // 1 block, and N threads
  fillArray<<<1,N>>>(dev_arr);

  cout << "------------ copy dev_arr to arr" << endl;
	// note the N * sizeof(int)
	hipMemcpy(arr, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);

  cout << "------------ printing changed host array" << endl;
	for(int i=0; i<N; i++)
	{
		printf("** changed host arr[%d] = %d\n", i, arr[i]);
	}

  // ---- FREE ALLOCATED KERNEL MEMORY
	hipFree( dev_arr );

  return 0;
}
